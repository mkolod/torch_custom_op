#include "hip/hip_runtime.h"
#include "saxpy_kernel.h"
#include <cstdio>

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

__global__ void saxpy_kernel(
  const float* const __restrict__ x,
  const float* const __restrict__ y,
  float* const __restrict__ output,
  const int numel,
  const float a) {

  const int gIdx = blockDim.x * blockIdx.x + threadIdx.x;
  if (gIdx > numel - 1) {
    return;
  }

  output[gIdx] = a * x[gIdx] + y[gIdx];
}

void saxpy_launcher(torch::Tensor* x,torch::Tensor* y, torch::Tensor* output, const int numel, const float a) {

  const int block = 1024;
  const int grid = (numel + block - 1) / block; 

  saxpy_kernel<<<grid, block>>>(x->data_ptr<float>(), y->data_ptr<float>(), output->data_ptr<float>(), numel, a);

   hipStreamSynchronize(0); 
   cudaCheckError();
}


