#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include "maxpool_2d_kernel.h"

__global__ void maxpool2d_kernel(
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const float* const __restrict__ X,
    float* const __restrict__ Y) {

  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int nc = blockIdx.x / Y_H;
  const int yh = blockIdx.x % Y_H;
  const float* X_ptr = X + nc * X_HxW;
  float* Y_ptr = Y + nc * Y_HxW;
  const int xh = yh * stride_h;
  const int t = max(xh - pad_t, 0);
  const int b = min(xh - pad_t + kernel_h, X_H);
  for (int yw = threadIdx.x; yw < Y_W; yw += blockDim.x) {
    const int xw = yw * stride_w;
    const int l = max(xw - pad_l, 0);
    const int r = min(xw - pad_l + kernel_w, X_W);
    float val = std::numeric_limits<float>::lowest();
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
#if __CUDA_ARCH__ >=350
        val = max(val, __ldg(X_ptr + i * X_W + j));
#else
        val = max(val, X_ptr[i * X_W + j]);
#endif
      }
    }
    Y_ptr[yh * Y_W + yw] = val;
  }
}

void maxpool2d_launcher(torch::Tensor input, torch::Tensor output, int height, int width, int outH, int outW, int kernel_size, int stride, int padding) {

  const dim3 threads(32, 32);
  const dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

  maxpool2d_kernel<<<blocks, threads>>>(height, width,
    outH, outW, kernel_size, kernel_size, stride, stride, padding, padding,
    input.data_ptr<float>(), output.data_ptr<float>());
  
  // check CUDA errors

}


